#include <cute/layout.hpp>
#include <cute/tensor.hpp>
#include <cute/algorithm/axpby.hpp>
#include <cutlass/gemm/device/gemm.h>
#include <hip/hip_runtime.h>
#include "../include/tensorimpl.cuh"

using namespace cutlass;

template <typename TensorType>
bool is_device_tensor(const TensorType& tensor) {
    return is_device_pointer(tensor.data());
}

int main(int argc, char* argv[]) {
    using dtype = float;

    hipStream_t stream;
    hipStreamCreate(&stream);

    // int M = 10;
    // int N = 10;
    // int K = 10;
    // auto a_ptr = new float[M * K];
    // auto b_ptr = new float[K * N];
    // auto c_ptr = new float[M * N];
    
    // for (int i = 0; i < M * K; ++i) {
    //     a_ptr[i] = 1;
    //     b_ptr[i] = 2;
    // }

    // auto a = cute::make_tensor(a_ptr, cute::make_shape(M, K), cute::make_stride(M, 1));
    // auto b = cute::make_tensor(b_ptr, cute::make_shape(K, N), cute::make_stride(K, 1));
    // auto c = cute::make_tensor(c_ptr, cute::make_shape(M, N), cute::make_stride(M, 1));
    // // cute::axpby(1, a, 1, b);
    // cute::print_tensor(b);

    auto tensor = TensorImpl<dtype>(2, 3);  // 2 x 3
    tensor.print_tensor(stream);

    auto size = 50;
    auto alloc_ptr = std::make_shared<DeviceAlloc<dtype>>(size);
    dtype src[size];
    for (int i = 0; i < size; ++i) {
        src[i] = (dtype)1.5;
    }
    alloc_ptr.get()->cpy_to_buffer(src, size, stream);
    dtype tmp[alloc_ptr.get()->size()];
    alloc_ptr.get()->cpy_from_buffer(tmp, stream);
    // for (int i = 0; i < size; ++i) {
        // std::cout << tmp[i] << " ";
    // }
    auto layout = Layout<Shape<int, int>, tuple<int, int>>(
        Shape<int, int>(5, 10),
        Stride<int, int>{10, Int<1>{}}
    );
    auto test = Tensor<ViewEngine<dtype*>, Layout<Shape<int, int>, tuple<int, int>>>((float*)tmp, layout);
    cute::print_tensor(test);
    // delete[] a_ptr;
    // delete[] b_ptr;
    // delete[] c_ptr;
    hipStreamDestroy(stream);
    std::cout << std::endl;
}
