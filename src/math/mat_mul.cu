#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "../utils.h"

static hipblasHandle_t cublas_handle;

__global__ void matmul_1(float* a, float* b, float* c) {
    
}

void matmul_forward(float *out, float* a, float* b,
                    int row, int inner, int col) {
    CUBLAS_CHECK(hipblasSgemm(
        cublas_handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        col, row, inner, &ALPHA,
        b, col, a, inner, &BETA,
        out, col
    ));
}
