#include <hipblas.h>
#include "math/mat_add.cu"
#include "math/mat_mul.cu"

int main(int argc, char** argv) {
    hipblasCreate(&cublas_handle);
    int config[5] = {3, 5, 10, 5, 2};
    int batch_size = 32;

    float* w1 = (float*)mallocCheck(sizeof(float) * config[0] * config[1]);  // (5, 3)
    float* w2 = (float*)mallocCheck(sizeof(float) * config[1] * config[2]);  // (10, 5)
    float* w3 = (float*)mallocCheck(sizeof(float) * config[2] * config[3]);  // (5, 10)
    float* w4 = (float*)mallocCheck(sizeof(float) * config[3] * config[4]);  // (2, 5)
    float* b1 = (float*)mallocCheck(sizeof(float) * config[1]);  // (5, 1)
    float* b2 = (float*)mallocCheck(sizeof(float) * config[2]);  // (10, 1)
    float* b3 = (float*)mallocCheck(sizeof(float) * config[3]);  // (5, 1)
    float* b4 = (float*)mallocCheck(sizeof(float) * config[4]);  // (2, 1)

    float* input = (float*)mallocCheck(sizeof(float) * batch_size * config[0]);  //  input shape (32, 3)
    // y = (x @ W^T) + b
    float* f1 = (float*)mallocCheck(sizeof(float) * batch_size * config[1]);
    float* f2 = (float*)mallocCheck(sizeof(float) * batch_size * config[2]);
    float* f3 = (float*)mallocCheck(sizeof(float) * batch_size * config[3]);
    float* f4 = (float*)mallocCheck(sizeof(float) * batch_size * config[4]);

    float* d_w1, *d_w2, *d_w3, *d_w4, *d_b1, *d_b2, *d_b3, *d_b4, *d_input, *d_f1, *d_f2, *d_f3, *d_f4;
    CUDA_CHECK(hipMalloc((void**)&d_w1, sizeof(float) * config[0] * config[1]));
    CUDA_CHECK(hipMalloc((void**)&d_w2, sizeof(float) * config[1] * config[2]));
    CUDA_CHECK(hipMalloc((void**)&d_w3, sizeof(float) * config[2] * config[3]));
    CUDA_CHECK(hipMalloc((void**)&d_w4, sizeof(float) * config[3] * config[4]));
    CUDA_CHECK(hipMalloc((void**)&d_b1, sizeof(float) * config[1]));
    CUDA_CHECK(hipMalloc((void**)&d_b2, sizeof(float) * config[2]));
    CUDA_CHECK(hipMalloc((void**)&d_b3, sizeof(float) * config[3]));
    CUDA_CHECK(hipMalloc((void**)&d_b4, sizeof(float) * config[4]));
    CUDA_CHECK(hipMalloc((void**)&d_input, sizeof(float) * batch_size * config[0]));
    CUDA_CHECK(hipMalloc((void**)&d_f1, sizeof(float) * batch_size * config[1]));
    CUDA_CHECK(hipMalloc((void**)&d_f2, sizeof(float) * batch_size * config[2]));
    CUDA_CHECK(hipMalloc((void**)&d_f3, sizeof(float) * batch_size * config[3]));
    CUDA_CHECK(hipMalloc((void**)&d_f4, sizeof(float) * batch_size * config[4]));

    CUDA_CHECK(hipFree(d_w1));
    CUDA_CHECK(hipFree(d_w2));
    CUDA_CHECK(hipFree(d_w3));
    CUDA_CHECK(hipFree(d_w4));
    CUDA_CHECK(hipFree(d_b1));
    CUDA_CHECK(hipFree(d_b2));
    CUDA_CHECK(hipFree(d_b3));
    CUDA_CHECK(hipFree(d_b4));
    CUDA_CHECK(hipFree(d_f1));
    CUDA_CHECK(hipFree(d_f2));
    CUDA_CHECK(hipFree(d_f3));
    CUDA_CHECK(hipFree(d_f4));
    hipblasDestroy(cublas_handle);
    free(w1);
    free(w2);
    free(w3);
    free(w4);
    free(b1);
    free(b2);
    free(b3);
    free(b4);
    return 0;
}
