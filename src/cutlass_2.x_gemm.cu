#include "cute/layout.hpp"
#include "cute/tensor.hpp"
#include "cutlass/gemm/device/gemm.h"

#include <hip/hip_runtime.h>
#include "../include/tensorimpl.cuh"
#include "../include/utils.h"

using namespace cutlass;
using namespace cute;

template <typename TensorType>
bool is_device_tensor(const TensorType& tensor) {
    return is_device_pointer(tensor.data());
}

int main(int argc, char* argv[]) {
    using dtype = float;
    hipStream_t stream;
    hipStreamCreate(&stream);
    int M = 3;
    int N = 5;
    int K = 6;
    int byte_size_a = sizeof(float) * M * K;
    int byte_size_b = sizeof(float) * K * N;
    int byte_size_c = sizeof(float) * M * N;
    float* A = new float[M * K];
    float* B = new float[K * N];
    float* C = new float[M * N];
    fill_increment<float>(A, M * K);
    fill_increment<float>(B, K * N);
    float *d_A;
    float *d_B;
    float *d_C;
    hipMallocAsync((void**)&d_A, byte_size_a, stream);
    hipMallocAsync((void**)&d_B, byte_size_b, stream);
    hipMallocAsync((void**)&d_C, byte_size_c, stream);
    hipMemcpyAsync(d_A, A, byte_size_a, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_B, B, byte_size_b, hipMemcpyHostToDevice, stream);
    using Gemm = cutlass::gemm::device::Gemm<
        float,
        cutlass::layout::RowMajor,
        float,
        cutlass::layout::RowMajor,
        float,
        cutlass::layout::RowMajor,
        float,
        cutlass::arch::OpClassSimt,
        cutlass::arch::Sm86
    >;
    Gemm gemm;
    Gemm::Arguments args(
        {M, N, K},
        {d_A, K},
        {d_B, N},
        {d_C, N},
        {d_C, N},
        {1.0f, 0.0f}
    );
    gemm(args, stream);
    hipMemcpyAsync(C, d_C, byte_size_c, hipMemcpyDeviceToHost, stream);
    hipDeviceSynchronize();
    for (int i = 0; i < M * N; ++i) {
        std::cout << C[i] << " ";
    }
    delete[] A;
    delete[] B;
    delete[] C;
    hipFreeAsync(d_A, stream);
    hipFreeAsync(d_B, stream);
    hipFreeAsync(d_C, stream);
    hipStreamDestroy(stream);
    std::cout << std::endl;
} 
