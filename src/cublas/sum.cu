#ifndef SUM
#define SUM

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "../utils.h"

void sum_cublas(float* sum, float* x, int size) {
    CUBLAS_CHECK(hipblasSasum(cublas_handle, size, x, 1, sum));
}

#endif