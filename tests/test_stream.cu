#include <string>
#include <hip/hip_runtime.h>

const int SIZE = 10000;
const int STREAMS = 3;

void test0() {
    float* host_ptrs_array[STREAMS] = {new float[SIZE], new float[SIZE], new float[SIZE]};
    float* dev_ptrs_arr[STREAMS];

    for (int i = 0; i < STREAMS; ++i) {
        hipMalloc(&dev_ptrs_arr[i], sizeof(float) * SIZE);
        hipMemcpy(dev_ptrs_arr[i], host_ptrs_array[i], sizeof(float) * SIZE, hipMemcpyHostToDevice);
    }
    for (int i = 0; i < STREAMS; ++i) {
        hipFree(dev_ptrs_arr[i]);
        delete[] host_ptrs_array[i];
    }
}

void test1() {
    hipStream_t stream_array[3];

    float* host_ptrs_arr[STREAMS];
    float* dev_ptrs_arr[STREAMS];

    for (int i = 0; i < STREAMS; ++i) {
        hipStreamCreate(&stream_array[i]);
    }
    for (int i = 0; i < STREAMS; ++i) {
        hipMallocAsync(&dev_ptrs_arr[i], sizeof(float) * SIZE, stream_array[i]);
        hipMemcpyAsync(dev_ptrs_arr[i], host_ptrs_arr[i], sizeof(float) * SIZE, hipMemcpyHostToDevice, stream_array[i]);
    }
    for (int i = 0; i < STREAMS; ++i) {
        hipFreeAsync(dev_ptrs_arr[i], stream_array[i]);
        hipHostFree(host_ptrs_arr[i]);
        hipStreamDestroy(stream_array[i]);
    }
}

int main(int argc, char** argv) {
    if (std::stoi(argv[1]) == 0) {
        test0();
    }
    else if (std::stoi(argv[1]) == 1) {
        test1();
    }
    printf("%s\n", argv[1]);
}
