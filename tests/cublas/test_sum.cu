#include <hipblas.h>
#include "../../src/utils.h"

void test0() {
    hipblasCreate(&cublas_handle);
    int size = 10;
    float* a = new float[size];
    fill_increment<float>(a, size);

    float* d_a;
    hipMalloc((void**)&d_a, sizeof(float) * size);
    hipMemcpy(d_a, a, sizeof(float) * size, hipMemcpyHostToDevice);

    float res;
    hipblasSasum(cublas_handle, size, d_a, 1, &res);
   
    hipblasDestroy(cublas_handle);
    delete[] a;
    hipFree(d_a);

    std::cout << res << std::endl;
}

int main(int argc, char** argv) {
    test0();
}