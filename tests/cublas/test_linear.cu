#include "../../include/utils.h"
#include "../../src/cublas/linear.cu"

void test_linear1() {
    hipblasCreate(&cublas_handle);
    float* weight = (float*)mallocCheck(sizeof(float) * 5 * 3);
    float* bias = (float*)mallocCheck(sizeof(float) * 5);
    float* input = (float*)mallocCheck(sizeof(float) * 3);
    float* fwd = (float*)mallocCheck(sizeof(float) * 5);

    fill_increment<float>(weight, 15);
    fill_ones<float>(bias, 5);
    fill_increment<float>(input, 3);
    fill_zeros<float>(fwd, 5);

    float* d_weight, *d_bias, *d_input, *d_fwd;
    CUDA_CHECK(hipMalloc((void**)&d_weight, sizeof(float) * 5 * 3));
    CUDA_CHECK(hipMalloc((void**)&d_bias, sizeof(float) * 5));
    CUDA_CHECK(hipMalloc((void**)&d_input, sizeof(float)* 3));
    CUDA_CHECK(hipMalloc((void**)&d_fwd, sizeof(float) * 5));

    CUDA_CHECK(hipMemcpy(d_weight, weight, sizeof(float) * 5 * 3, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_bias, bias, sizeof(float) * 5, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_input, input, sizeof(float) * 3, hipMemcpyHostToDevice));

    linear_forward(d_fwd, d_weight, d_input, d_bias, 1, 3, 5);

    CUDA_CHECK(hipMemcpy(fwd, d_fwd, sizeof(float) * 5, hipMemcpyDeviceToHost));

    print_matrix(1, 5, fwd, 5);
    // expected: [14, 32, 50, 68, 86] + [1, 1, 1, 1, 1] = [15, 33, 51, 69, 87]

    CUDA_CHECK(hipFree(d_weight));
    CUDA_CHECK(hipFree(d_bias));
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_fwd));
    free(weight);
    free(bias);
    free(input);
    free(fwd);
    hipblasDestroy(cublas_handle);
}

int main(int argc, char** argv)
{
    test_linear1();
}
