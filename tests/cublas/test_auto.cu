#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "../../include/utils.h"
#include "../../src/cublas/add.cu"
#include "../../src/cublas/sum.cu"

int main(int argc, char** argv) { 
    CUBLAS_CHECK(hipblasCreate(&cublas_handle));  // don't declare cublas_handle; create the static instance from utils.h
    // hipblasSetPointerMode(cublas_handle, HIPBLAS_POINTER_MODE_DEVICE); 
    int size = 10;
    int streams = 2;
    hipStream_t stream_arr[streams];

    CUDA_CHECK(hipStreamCreate(&stream_arr[0]));
    CUDA_CHECK(hipStreamCreate(&stream_arr[1]));
    // TODO: wrap the rest of these in CUDA_CHECK macro
    float *a, *b, *c, *d;
    hipHostMalloc((void**)&a, sizeof(float) * size);
    hipHostMalloc((void**)&b, sizeof(float) * size);
    hipHostMalloc((void**)&c, sizeof(float) * size);
    hipHostMalloc((void**)&d, sizeof(float));
    fill_increment<float>(a, size);
    fill_increment<float>(b, size);

    float *d_a, *d_b, *d_c, *d_d;
    hipMallocAsync((void**)&d_a, sizeof(float) * size, stream_arr[0]);
    hipMallocAsync((void**)&d_b, sizeof(float) * size, stream_arr[0]);
    hipMallocAsync((void**)&d_c, sizeof(float) * size, stream_arr[1]);
    // hipMallocAsync((void**)&d_d, sizeof(float), stream_arr[1]);
    hipMemcpyAsync(d_a, a, sizeof(float) * size, hipMemcpyHostToDevice, stream_arr[0]);
    hipMemcpyAsync(d_b, b, sizeof(float) * size, hipMemcpyHostToDevice, stream_arr[0]);

    // hipDeviceSynchronize();  // do we need this?

    // "forward" pass
    add_cublas(d_c, d_a, d_b, 1, size);
    sum_cublas(d, d_c, size);
    
    // hipMemcpy(d, d_d, sizeof(float), hipMemcpyDeviceToHost);
    std::cout << *d << std::endl;  // 110 for size=10
    
    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);
    hipHostFree(d);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    // hipFree(d_d);
    CUBLAS_CHECK(hipblasDestroy(cublas_handle));
    return 0;
}