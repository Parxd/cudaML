#include <string>
#include <hip/hip_runtime.h>

const int SIZE = 10000;
const int STREAMS = 3;

void test0() {
    // synchronous HtoD memory
    float* host_ptrs_array[STREAMS] = {new float[SIZE], new float[SIZE], new float[SIZE]};
    float* dev_ptrs_arr[STREAMS];

    for (int i = 0; i < STREAMS; ++i) {
        hipMalloc(&dev_ptrs_arr[i], sizeof(float) * SIZE);
        hipMemcpy(dev_ptrs_arr[i], host_ptrs_array[i], sizeof(float) * SIZE, hipMemcpyHostToDevice);
    }
    for (int i = 0; i < STREAMS; ++i) {
        hipFree(dev_ptrs_arr[i]);
        delete[] host_ptrs_array[i];
    }
}

void test1() {
    // asynchronous HtoD memcpy
    hipStream_t stream_array[3];

    float* host_ptrs_arr[STREAMS];
    float* dev_ptrs_arr[STREAMS];

    for (int i = 0; i < STREAMS; ++i) {
        hipHostMalloc((void**)&host_ptrs_arr[i], sizeof(float) * SIZE, hipHostMallocDefault);
        hipStreamCreate(&stream_array[i]);
    }
    for (int i = 0; i < STREAMS; ++i) {
        hipMallocAsync(&dev_ptrs_arr[i], sizeof(float) * SIZE, stream_array[i]);
        hipMemcpyAsync(dev_ptrs_arr[i], host_ptrs_arr[i], sizeof(float) * SIZE, hipMemcpyHostToDevice, stream_array[i]);
    }
    for (int i = 0; i < STREAMS; ++i) {
        hipFreeAsync(dev_ptrs_arr[i], stream_array[i]);
        hipHostFree(host_ptrs_arr[i]);
        hipStreamDestroy(stream_array[i]);
    }
}

int main(int argc, char** argv) {
    if (std::stoi(argv[1]) == 0) {
        test0();
    }
    else if (std::stoi(argv[1]) == 1) {
        test1();
    }
    printf("%s\n", argv[1]);
    // ideally, ./test_stream 1 should execute faster than ./test_stream 0

    // $ nsys profile ./test_stream 0
    // $ nsys profile ./test_stream 1
}
