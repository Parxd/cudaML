#include <iostream>
#include <hipblas.h>
#include "../../src/utils.h"

void test1() {

}

void test2() {
    
}

void test_cublas1() {
    hipblasHandle_t handle = NULL;
    hipblasCreate(&handle);
    
    const float alpha = 1.0;
    const float beta = 0.0;
    const int m = 2;
    const int n = 3;
    const int k = 2;
    
    auto a = new float[m * k];
    auto b = new float[k * n];
    auto c = new float[m * n];

    fill_increment<float>(a, m * k);
    fill_increment<float>(b, k * n);
    // A: m x k
    // B: k x n

    // B^T: n * k
    // A^T: k * m

    // A @ B = C
    // B^T @ A^T = C^T -- row-major
    // B @ A = C -- column major
    float *d_a, *d_b, *d_c;
    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_a), sizeof(float) * m * k));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_b), sizeof(float) * k * n));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&d_c), sizeof(float) * m * n));

    CUDA_CHECK(hipMemcpy(d_a, a, sizeof(float) * m * k, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, b, sizeof(float) * k * n, hipMemcpyHostToDevice));

    // input leading dims as if it was column-major...
    // switch operands & transpose second matrix
    CUBLAS_CHECK(hipblasSgemm(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        n, m, k, &alpha,
        d_b, n, d_a, k,
        &beta, d_c, n
    ));
    CUDA_CHECK(hipMemcpy(c, d_c, sizeof(float) * m * n, hipMemcpyDeviceToHost));

    print_matrix(m, n, c, n);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    delete[] a;
    delete[] b;
    delete[] c;
    
    hipblasDestroy(handle);
    hipDeviceReset();
}

int main(int argc, char** argv) {
    // test1();
    test_cublas1();
}
