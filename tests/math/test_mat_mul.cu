#include <iostream>
#include <hipblas.h>
#include "../../src/utils.h"

void test1() {

}

void test2() {
    
}

void test_cublas1() {
    hipblasHandle_t handle = NULL;
    hipStream_t stream = NULL;
    hipblasCreate(&handle);
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipblasSetStream(handle, stream);
    
    const float alpha = 1.0;
    const float beta = 0.0;
    const int m = 2;
    const int n = 2;
    const int k = 2;

    auto a = new float[m * k];
    auto b = new float[k * n];
    auto c = new float[m * n];

    fill_increment<float>(a, m * k);
    fill_increment<float>(b, k * n);

    float *d_a, *d_b, *d_c;
    hipMalloc(reinterpret_cast<void**>(&d_a), sizeof(float) * m * k);
    hipMalloc(reinterpret_cast<void**>(&d_b), sizeof(float) * k * n);
    hipMalloc(reinterpret_cast<void**>(&d_c), sizeof(float) * m * n);

    hipMemcpy(d_a, a, sizeof(float) * m * k, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * k * n, hipMemcpyHostToDevice);
    
    hipblasSgemm(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        m, n, k, &alpha,
        d_a, k, d_b, n,
        &beta, d_c, n
    );
    hipMemcpy(c, d_c, sizeof(float) * m * n, hipMemcpyDeviceToHost);

    print_matrix(m, n, c, n);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    delete[] a;
    delete[] b;
    delete[] c;
}

int main(int argc, char** argv) {
    test1();
    test_cublas1();
}
