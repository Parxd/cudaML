#include "../../src/utils.h"
#include "../../src/math/vec_add.cu"
#include "../../src/math/mat_add.cu"


static int N = 2;
static int M = 3;


int main(int argc, char** argv) {
    int byte_size = N * M * sizeof(float);

    float *a, *b, *c;
    a = (float*)malloc(byte_size);
    b = (float*)malloc(byte_size);
    c = (float*)malloc(byte_size);
    
    fill_ones(a, N * M);
    fill_ones(b, N * M);

    float *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, byte_size);
    hipMalloc((void**)&d_b, byte_size);
    hipMalloc((void**)&d_c, byte_size);
    
    hipMemcpy(d_a, a, byte_size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, byte_size, hipMemcpyHostToDevice);

    // with 1D block:
    // vecadd_2<<<1, N * M>>>(d_a, d_b, d_c);

    // with 2D block:
    // need 1 block of shape 2 x 3
    dim3 gridDim(1, 1, 1);
    dim3 blockDim(N, M, 1);
    matadd_2<<<gridDim, blockDim>>>(d_a, d_b, d_c, N, M);

    hipMemcpy(c, d_c, byte_size, hipMemcpyDeviceToHost);

    print_matrix(c, N, M);

    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}
