#include "../../src/utils.h"
#include "../../src/math/mat_add.cu"

void test1() {
    int N = 2;
    int M = 3;
    int byte_size = N * M * sizeof(float);

    float *a, *b, *c;
    a = (float*)malloc(byte_size);
    b = (float*)malloc(byte_size);
    c = (float*)malloc(byte_size);
    
    fill_ones(a, N * M);
    fill_ones(b, N * M);

    float *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, byte_size);
    hipMalloc((void**)&d_b, byte_size);
    hipMalloc((void**)&d_c, byte_size);
    
    hipMemcpy(d_a, a, byte_size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, byte_size, hipMemcpyHostToDevice);

    // with 1D block:
    // vecadd_2<<<1, N * M>>>(d_a, d_b, d_c);

    // with 2D block:
    // need 1 block of shape 2 x 3
    dim3 gridDim(1, 1, 1);
    dim3 blockDim(N, M, 1);
    matadd_1<<<gridDim, blockDim>>>(d_a, d_b, d_c, N, M);

    hipMemcpy(c, d_c, byte_size, hipMemcpyDeviceToHost);

    print_matrix(c, N, M);

    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

void test2() {
    int N = 41;
    int M = 37;
    int byte_size = N * M * sizeof(float);

    float *a, *b, *c;
    a = (float*)malloc(byte_size);
    b = (float*)malloc(byte_size);
    c = (float*)malloc(byte_size);
    fill_ones(a, N);
    fill_ones(b, N);

    float *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, byte_size);
    hipMalloc((void**)&d_b, byte_size);
    hipMalloc((void**)&d_c, byte_size);
    
    hipMemcpy(d_a, a, byte_size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, byte_size, hipMemcpyHostToDevice);

    dim3 gridDim(CEIL_DIV(N, MAX_THREADS), CEIL_DIV(M, MAX_THREADS));
    dim3 blockDim(MAX_THREADS, MAX_THREADS, 1);
    matadd_2<<<gridDim, blockDim>>>(d_a, d_b, d_c);
}

int main(int argc, char** argv) {
    // test1();
    test2();
}
